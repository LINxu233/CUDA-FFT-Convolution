#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "cudaConvFFTData.h"
#include "cudaConvFFTData.cuh"

static bool debug = false;

enum IN_INDEX{
    DATA_INDEX,
    KERNEL_H_INDEX,
    KERNEL_W_INDEX
};
////////////////////////////////////////////////////////////////////////////////
// Mex Entry
////////////////////////////////////////////////////////////////////////////////
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare all variables.*/
    const mxArray *mxDATA = prhs[DATA_INDEX];
    mxGPUArray *FFT_DATA;
    float2 *d_CFFT_DATA;
    float *h_Data;
    float *d_Data;
    float *d_PaddedData;
    char const * const errId = "parallel:gpu:mexGPUExample:InvalidInput";
    char const * const errMsg = "Invalid input to MEX file.";

    /* Choose a reasonably sized number of threads for the block. */
    int const THREAD_PER_BLOCK_H = 16;
    int const THREAD_PER_BLOCK_W = 8;
    int const THREAD_PER_BLOCK_D = 8;

    // int MblocksPerGrid, NblocksPerGrid;
    int KERNEL_H, KERNEL_W, DATA_H, DATA_W, 
        PADDING_H, PADDING_W, FFT_H, FFT_W, FEATURE_DIM,
        DATA_SIZE, FFT_SIZE, CFFT_SIZE;

    
    /* Initialize the MathWorks GPU API. */
    // If initialized mxInitGPU do nothing
    if (mxInitGPU() != MX_GPU_SUCCESS)
        mexErrMsgTxt("mxInitGPU fail");

    
    /* Throw an error if the input is not a GPU array. */
    if ((nrhs!=3) ||
            mxIsGPUArray(mxDATA) || 
            mxGetNumberOfDimensions(mxDATA) != 3 || 
            mxGetClassID(mxDATA) != mxSINGLE_CLASS) {
        mexErrMsgIdAndTxt(errId, errMsg);
    }


    // Kernel dimensions
    KERNEL_H = (int)mxGetScalar(prhs[KERNEL_H_INDEX]);
    KERNEL_W = (int)mxGetScalar(prhs[KERNEL_W_INDEX]);
    if(debug) fprintf(stderr,"Kernel size: h=%d, w=%d\n",KERNEL_H,KERNEL_W);

    // Data dimensions
    const mwSize *DATA_dims = mxGetDimensions(mxDATA);
    DATA_H = DATA_dims[0];
    DATA_W = DATA_dims[1];
    FEATURE_DIM = DATA_dims[2];

    h_Data = (float *)mxGetData(mxDATA);
    if(debug) fprintf(stderr,"Data size: h=%d, w=%d, f=%d\n",DATA_H,DATA_W,FEATURE_DIM); 

    // Width and height of padding
    PADDING_H = KERNEL_H - 1;
    PADDING_W = KERNEL_W - 1;

    // Derive FFT size from data and kernel dimensions
    // FFT_H = computeFFTsize(DATA_H + PADDING_H);
    // FFT_W = computeFFTsize(DATA_W + PADDING_W);
    FFT_H = computeFFTsize16(DATA_H + PADDING_H);
    FFT_W = computeFFTsize16(DATA_W + PADDING_W);

    if(debug) fprintf(stderr,"FFT size: h=%d, w=%d\n",FFT_H,FFT_W);

    DATA_SIZE = DATA_W * DATA_H * FEATURE_DIM * sizeof(float);
    FFT_SIZE  = FFT_W  * FFT_H  * FEATURE_DIM * sizeof(float);
    // CFFT_SIZE = FFT_W  * FFT_H  * FEATURE_DIM * sizeof(float2);

    // Allocate memory for input
    // No need to initialize using mxCalloc

    mwSize CFFT_dims[3];

    CFFT_dims[0] = FFT_H/2 + 1;
    CFFT_dims[1] = FFT_W;
    CFFT_dims[2] = FEATURE_DIM;

    /* Wrap the result up as a MATLAB gpuArray for return. */
    FFT_DATA = mxGPUCreateGPUArray(3,
                                CFFT_dims,
                                mxSINGLE_CLASS,
                                mxCOMPLEX,
                                MX_GPU_INITIALIZE_VALUES);
    
    d_CFFT_DATA = (float2 *)mxGPUGetData(FFT_DATA);
    
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_Data,        DATA_SIZE));
    CUDA_SAFE_CALL_NO_SYNC(hipMalloc((void **)&d_PaddedData,  FFT_SIZE));

    CUDA_SAFE_CALL_NO_SYNC(hipMemcpy(d_Data, h_Data, DATA_SIZE, hipMemcpyHostToDevice));

    dim3 threadBlock(THREAD_PER_BLOCK_H, THREAD_PER_BLOCK_W, THREAD_PER_BLOCK_D);
    dim3 dataBlockGrid( iDivUp(FFT_W, threadBlock.x), 
                        iDivUp(FFT_H, threadBlock.y), 
                        iDivUp(FEATURE_DIM, threadBlock.z));

    padData<<<dataBlockGrid, threadBlock>>>(
            d_PaddedData,
            d_Data,
            FFT_W,
            FFT_H,
            DATA_W,
            DATA_H,
            FEATURE_DIM
        );

    if(debug) fprintf(stderr,"Padding\n");

    int BATCH = FEATURE_DIM;
    int FFT_Dims[] = { FFT_W, FFT_H };

    int idist = FFT_W * FFT_H;
    int odist = FFT_W * (FFT_H/2 + 1);
    
    int inembed[] = {FFT_W, FFT_H};
    int onembed[] = {FFT_W, FFT_H/2 + 1};

    hipfftHandle FFTplan_R2C;
    CUFFT_SAFE_CALL(hipfftPlanMany(&FFTplan_R2C, 
        2, // rank
        FFT_Dims, 
        inembed, 1, idist, // *inembed, istride, idist
        onembed, 1, odist, // *onembed, ostride, odist
        HIPFFT_R2C, 
        BATCH)); // batch


    CUFFT_SAFE_CALL(hipfftExecR2C(FFTplan_R2C, d_PaddedData, d_CFFT_DATA));
    CUDA_SAFE_CALL_NO_SYNC(hipDeviceSynchronize());
    if(debug) fprintf(stderr,"Sync\n");

    plhs[0] = mxGPUCreateMxArrayOnGPU(FFT_DATA);
    if(debug) fprintf(stderr,"plhs\n");
    /*
     * The mxGPUArray pointers are host-side structures that refer to device
     * data. These must be destroyed before leaving the MEX function.
     */
    mxGPUDestroyGPUArray(FFT_DATA);
    hipfftDestroy(FFTplan_R2C);
    hipFree(d_Data);
    hipFree(d_PaddedData);
}
